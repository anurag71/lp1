#include<stdio.h>
#include<hip/hip_runtime.h>


#define SIZE 10

__global__ void min(int *input){
	int tid = threadIdx.x;
	int step_size=1;
	int numberofthreads = blockDim.x;
	while(numberofthreads>0){
		if(tid<numberofthreads){
			int first = tid*step_size*2;
			int second = first+step_size;
			if(input[second]<input[first])
				input[first]=input[second];
		}
			step_size*=2;
			numberofthreads/=2; 
	}
}

int main()
{
	// printf("Enter size of array:");
	// scanf("%d",&SIZE);
	int arr[SIZE];

	int byte_size = SIZE*sizeof(int);

	for(int i=0;i<SIZE;i++){
		arr[i] = rand()% 100;
	}

	printf("The array is:\n");
	for(int i=0;i<SIZE;i++){
		printf("%d ",arr[i]=rand());
	}
	printf("\n");

	int *arr_min,result;
	hipMalloc(&arr_min,byte_size);
	hipMemcpy(arr_min,arr,byte_size,hipMemcpyHostToDevice);
	min<<<1,SIZE/2>>>(arr_min);
	hipMemcpy(&result,arr_min,sizeof(int),hipMemcpyDeviceToHost);
	printf("Minimum: %d", result);

	return 0;
}