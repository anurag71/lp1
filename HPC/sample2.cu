#include<iostream>
#include<hip/hip_runtime.h>


#define SIZE 9

using namespace std;

__global__ void vectoradd(int *p,int *q,int *result){

	int tid = threadIdx.x + blockDim.x * blockIdx.x;
	if(tid<SIZE){
		// for(int i=0;i<SIZE;i++){
			result[tid] = p[tid] + q[tid];
		// }
	}
}

int main(int argc, char const *argv[]){
		
	int v1[SIZE],v2[SIZE],z[SIZE];
	for(int i=0;i<SIZE;i++){
		v1[i] = rand()%100+1;
        v2[i] = rand()%50+1;
        z[i] = 0;
	}

	printf("First Vector:\n");
	for(int i=0;i<SIZE;i++){
		printf("%d ",v1[1]);
	}
	printf("\nSecond Vector:\n");
	for(int i=0;i<SIZE;i++){
		printf("%d ",v2[1]);
	}

	int byte_size = SIZE * sizeof(int);

	int *a,*b,*c;
	hipMalloc(&a,byte_size);
	hipMalloc(&b,byte_size);
	hipMalloc(&c,byte_size);
	hipMemcpy(a,v1,byte_size,hipMemcpyHostToDevice);
	hipMemcpy(b,v2,byte_size,hipMemcpyHostToDevice);
	hipMemcpy(c,z,byte_size,hipMemcpyHostToDevice);
	vectoradd<<<2,SIZE>>>(a,b,c);
	hipMemcpy(&z,c,byte_size,hipMemcpyDeviceToHost);

	printf("\nResult:\n");
	for(int i=0;i<SIZE;i++){
		printf("%d ",z[1]);
	}

	return 0;
}