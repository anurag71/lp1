
#include <hip/hip_runtime.h>
#include<iostream>
#include<ctime>
#include<cmath>
#include<stdexcept>

using namespace std;

#define MaxElement 1000

__global__ void Sum(int* Array1, int* Array2, int* Result, int ElementCount){
	int Index = blockIdx.x * blockDim.x + threadIdx.x;

	if(Index < ElementCount)
		Result[Index] = Array1[Index] + Array2[Index];
}

void HostVectorSum(int ArraySize=1000, int ThreadsPerBlock=100){
	int ArrayMemory = ArraySize * sizeof(int);

	int* HostArray1 = (int*) malloc(ArrayMemory);
	int* HostArray2 = (int*) malloc(ArrayMemory);
	int* HostResult = (int*) malloc(ArrayMemory);

	int* DeviceArray1;
	int* DeviceArray2;
	int* DeviceResult;

	srand(time(0));

	for(int i=0;i<ArraySize;i++){
		HostArray1[i] = rand() % MaxElement;
		HostArray2[i] = rand() % MaxElement;
	}

	hipMalloc(&DeviceArray1, ArrayMemory);
	hipMalloc(&DeviceArray2, ArrayMemory);
	hipMalloc(&DeviceResult, ArrayMemory);

	hipMemcpy(DeviceArray1, HostArray1, ArrayMemory, hipMemcpyHostToDevice);
	hipMemcpy(DeviceArray2, HostArray2, ArrayMemory, hipMemcpyHostToDevice);
	
	int BlocksPerGrid = 1;

	if(ArraySize > ThreadsPerBlock)
		BlocksPerGrid = ceil(double(ArraySize) / double(ThreadsPerBlock));

	Sum<<<BlocksPerGrid, ThreadsPerBlock>>>(DeviceArray1, DeviceArray2, DeviceResult, ArraySize);
	
	hipMemcpy(HostResult, DeviceResult, ArrayMemory, hipMemcpyDeviceToHost);

	hipFree(DeviceArray1);
	hipFree(DeviceArray2);
	hipFree(DeviceResult);

	for(int i=0;i<ArraySize;i++)
		printf("Index %d --> %d + %d = %d\n", i+1, HostArray1[i], HostArray2[i], HostResult[i]);

	free(HostArray1);
	free(HostArray2);
	free(HostResult);
}

__global__ void VectorMatrixMultiplication(int* Vector, int* Matrix, int* Result, int Row, int Column){
	int Index = blockIdx.x * blockDim.x + threadIdx.x;

	int Sum = 0;

	if(Index < Column){
		int ColumnStartIndex = Index * Row;
		for(int i=0;i<Row;i++)
			Sum += Vector[i] * Matrix[ColumnStartIndex + i];

		Result[Index] = Sum;
	}
}

void HostVectorMatrixMultiplication(int Row, int Column){
	int* HostArray = (int*) malloc(Row * sizeof(int));
	int* HostMatrix = (int*) malloc(Row * Column * sizeof(int));
	int* HostResult = (int*) malloc(Column * sizeof(int));

	int* DeviceArray;
	int* DeviceMatrix;
	int* DeviceResult;

	srand(time(0));

	for(int i=0;i<Row;i++)
		HostArray[i] = rand() % MaxElement;

	for(int i=0;i<Column;i++)
		for(int j=0;j<Row;j++)
			HostMatrix[i*Row+j] = rand() % MaxElement;

	hipMalloc(&DeviceArray, Row*sizeof(int));
	hipMalloc(&DeviceMatrix, Row*Column*sizeof(int));
	hipMalloc(&DeviceResult, Column*sizeof(int));

	hipMemcpy(DeviceArray, HostArray, Row*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(DeviceMatrix, HostMatrix, Row*Column*sizeof(int), hipMemcpyHostToDevice);

	VectorMatrixMultiplication<<<Column, 1>>>(DeviceArray, DeviceMatrix, DeviceResult, Row, Column);

	hipMemcpy(HostResult, DeviceResult, Column*sizeof(int), hipMemcpyDeviceToHost);

	hipFree(DeviceArray);
	hipFree(DeviceMatrix);
	hipFree(DeviceResult);

	for(int i=0;i<Column;i++)
		printf("Index %d --> %d\n", i+1,HostResult[i]);

	free(HostArray);
	free(HostMatrix);
	free(HostResult);
}

__global__ void MatrixMultiplication(int* MatrixA, int* MatrixB, int* Result, int Dimension){
	int Row = blockIdx.y * blockDim.y + threadIdx.y;
	int Column = blockIdx.x * blockDim.x + threadIdx.x;

	int Sum = 0;
	if(Row < Dimension && Column < Dimension){
		for(int i=0;i<Dimension;i++)
			Sum += MatrixA[Row * Dimension + i] * MatrixB[i * Dimension + Column];
		__syncthreads();
		Result[Row * Dimension + Column] = Sum;
	}
}

void HostMatrixMultiplication(int Dimension){
	int MatrixMemory = Dimension * Dimension * sizeof(int);

	int* HostMatrixA = (int*) malloc(MatrixMemory);
	int* HostMatrixB = (int*) malloc(MatrixMemory);
	int* HostResult = (int*) malloc(MatrixMemory);

	srand(time(0));

	for(int i=0;i<Dimension;i++){
		for(int j=0;j<Dimension;j++){
			HostMatrixA[i * Dimension + j] = rand() % 30;
			HostMatrixB[i * Dimension + j] = rand() % 30;
		}
	}

	int* DeviceMatrixA;
	int* DeviceMatrixB;
	int* DeviceResult;

	hipMalloc(&DeviceMatrixA, MatrixMemory);
	hipMalloc(&DeviceMatrixB, MatrixMemory);
	hipMalloc(&DeviceResult, MatrixMemory);

	hipMemcpy(DeviceMatrixA, HostMatrixA, MatrixMemory, hipMemcpyHostToDevice);
	hipMemcpy(DeviceMatrixB, HostMatrixB, MatrixMemory, hipMemcpyHostToDevice);

	dim3 ThreadsPerBlock(Dimension, Dimension);
	dim3 BlocksPerGrid(1, 1);

	MatrixMultiplication<<<BlocksPerGrid, ThreadsPerBlock>>>(DeviceMatrixA, DeviceMatrixB, DeviceResult, Dimension);

	hipError_t Exception = hipGetLastError();

	if(Exception != hipSuccess){
		printf("Cuda Error: %s", hipGetErrorString(Exception));
		return;
	}

	hipDeviceSynchronize();

	hipMemcpy(HostResult, DeviceResult, MatrixMemory, hipMemcpyDeviceToHost);

	hipFree(DeviceMatrixA);
	hipFree(DeviceMatrixB);
	hipFree(DeviceResult);

	for(int i=0;i<Dimension;i++){
		for(int j=0;j<Dimension;j++){
			printf("%d ", HostResult[i * Dimension + j]);
		}
		printf("\n");
	}
}

int main(){
	int Choice;

	printf("1.Vector Addition\n2.Vector Matrix Multiplication\n3.Matrix Multiplication\n4.Exit\n");
	printf("Enter The Operation To Be Performed: : ");
	scanf("%d", &Choice);

	if(Choice==1){
		int ArraySize;
		printf("Enter The Array Size: : ");
		scanf("%d", &ArraySize);
		HostVectorSum(ArraySize);
	}
	else if(Choice==2){
		int Row, Column;
		printf("Enter The Rows And Columns Of The Matrix: : ");
		scanf("%d %d", &Row, &Column);
		HostVectorMatrixMultiplication(Row, Column);
	}
	else if(Choice==3){
		int Dimension;
		printf("Enter The Dimensions Of The Matrix: : ");
		scanf("%d", &Dimension);
		HostMatrixMultiplication(Dimension);
	}
	else
		return 0;
	
	return 0;
}
